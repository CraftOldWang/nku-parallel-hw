#include "guessing_cuda.h"
#include "PCFG.h"


#define CUDA_CHECK(call) \
    do { \
        hipError_t err__ = (call); \
        if (err__ != hipSuccess) { \
            fprintf(stderr, "\033[1;31m[CUDA ERROR]\033[0m %s:%d: %s (%d)\n", \
                __FILE__, __LINE__, hipGetErrorString(err__), err__); \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

// 安全的CUDA错误检查（用于回调函数）
#define CUDA_CHECK_CALLBACK(call, cleanup_action) \
    do { \
        hipError_t err__ = (call); \
        if (err__ != hipSuccess) { \
            std::cerr << "CUDA error in callback: " << hipGetErrorString(err__) << std::endl; \
            cleanup_action; \
            return; \
        } \
    } while (0)

AsyncGpuPipeline::AsyncTaskData::AsyncTaskData(TaskManager&& tm)
    : task_manager(std::move(tm)), gpu_buffer(nullptr) {
    // 创建CUDA streams和events
    hipStreamCreate(&compute_stream);

    // 初始化指针
    temp_prefixs = nullptr;
    d_seg_types = nullptr;
    d_seg_ids = nullptr;
    d_seg_lens = nullptr;
    d_prefix_offsets = nullptr;
    d_prefix_lens = nullptr;
    d_seg_value_counts = nullptr;
    d_cumulative_guess_offsets = nullptr;
    d_output_offsets = nullptr;
    d_tasks = nullptr;
    d_guess_buffer = nullptr;
    h_prefix_offsets = nullptr;
    result_len = 0;
}

AsyncGpuPipeline::AsyncTaskData::~AsyncTaskData(){
            // 清理CUDA资源
            if (compute_stream) hipStreamDestroy(compute_stream);
}



void sync_gpu_task(AsyncGpuTask* task_data, PriorityQueue& q) {
#ifdef DEBUG
    printf("[DEBUG] 🎯 sync_gpu_task: Starting synchronous GPU task\n");
#endif

    try {
        
        // 创建临时数据结构
        AsyncGpuPipeline::AsyncTaskData data(std::move(task_data->task_manager));
        
        // 阶段1: 准备GPU数据
        prepare_gpu_data_stage(data);

        
        // 阶段2: 启动GPU kernel
        launch_kernel_stage(data);
        
        // 🔥 阶段3: 同步等待GPU完成
        hipError_t sync_err = hipStreamSynchronize(data.compute_stream);
        if (sync_err != hipSuccess) {
            throw std::runtime_error("GPU kernel execution failed: " + std::string(hipGetErrorString(sync_err)));
        }
        
        // 阶段4: 同步拷贝结果回CPU
        CUDA_CHECK(hipMemcpy(data.gpu_buffer, data.d_guess_buffer, 
                             data.result_len * sizeof(char), hipMemcpyDeviceToHost));
        
        // 阶段5: CPU处理
        process_strings_stage(data);
        
        // 阶段6: 合并结果到全局队列
        merge_results_stage(data);
        
        // 阶段7: 同步清理所有资源
        synchronous_cleanup(data);
        
#ifdef DEBUG
        printf("[DEBUG] ✅ sync_gpu_task: GPU task completed successfully\n");
#endif

    } catch (const std::exception& e) {
        std::cerr << "Sync GPU task failed: " << e.what() << std::endl;
        // 异常会在析构函数中清理资源
    }
    
    delete task_data;

#ifdef DEBUG
    printf("[DEBUG] 🗑️ sync_gpu_task: Task data deleted\n");
#endif
}


void AsyncGpuPipeline::launch_async_pipeline(TaskManager tm, PriorityQueue& q) {

    auto* async_data = new AsyncTaskData(std::move(tm));

    try {
        // 阶段1: 准备数据（同步，但可以优化）
        prepare_gpu_data_stage(*async_data);

        // 阶段2: 异步启动GPU kernel
        launch_kernel_stage(*async_data);

        // 设置kernel完成回调

        // 阶段3: 启动内存拷贝+注册回调（异步）→ 立即返回(这个立即返回埋下了伏笔)
        // start_memory_copy_stage(*async_data);
        // 🔥 修改的阶段3: 把内存拷贝任务提交到线程池
        submit_memory_copy_task(async_data, q);

#ifdef DEBUG
        printf("[DEBUG] ✅ Async pipeline launched successfully\n");
#endif

    } catch (const std::exception& e) {
#ifdef DEBUG
        printf("[DEBUG] ❌ Failed to launch async pipeline: %s\n", e.what());
#endif
        std::cerr << "Failed to launch async pipeline: " << e.what() << std::endl;
        delete async_data;
    }
}



// 🔥 新函数：提交内存拷贝任务到线程池
void submit_memory_copy_task(AsyncGpuPipeline::AsyncTaskData* data, PriorityQueue& q) {
#ifdef DEBUG
    printf("[DEBUG] 📤 submit_memory_copy_task: Submitting memory copy task to thread pool\n");
#endif

#ifdef DEBUG
            printf("[DEBUG] ✅ start launch async  copying results back\n");
#endif
    // 🔥 关键：使用同一个 compute_stream，自动等待kernel完成
    // 这里异步一下， 然后 提交的任务也是异步， 然后接收任务的 线程就需要等待了。
    hipError_t err = hipMemcpyAsync(data->gpu_buffer, data->d_guess_buffer,
                                     data->result_len * sizeof(char),
                                     hipMemcpyDeviceToHost, data->compute_stream);
    if (err != hipSuccess) {
        std::cerr << "Memory copy scheduling failed: " << hipGetErrorString(err) << std::endl;

        return;
    }

#ifdef DEBUG
            printf("[DEBUG] ✅ launch async  copying results back succesfully\n");
#endif
    // 🔥 关键：提交到线程池，受到MAX_PENDING_TASKS限制
    thread_pool->enqueue([data, &q]() {
        try {
#ifdef DEBUG
            printf("[DEBUG] 🧵 Memory copy task started - waiting for GPU...\n");
#endif
            
            // 🔥 等待GPU kernel完成（同步点）
            hipError_t sync_err = hipStreamSynchronize(data->compute_stream);
            if (sync_err != hipSuccess) {
                throw std::runtime_error("GPU kernel execution failed: " + std::string(hipGetErrorString(sync_err)));
            }

#ifdef DEBUG
            printf("[DEBUG] ✅ Memory copy completed, processing results\n");
#endif
            

            
            // CPU处理阶段
            process_strings_stage(*data);
            
            // 合并结果
            merge_results_stage(*data);
            
            // 清理资源
            synchronous_cleanup(*data);

#ifdef DEBUG
            printf("[DEBUG] ✅ Memory copy task completed successfully\n");
#endif

        } catch (const std::exception& e) {
            std::cerr << "Memory copy task failed: " << e.what() << std::endl;
            data->has_error = true;
            
            // 错误时也要清理
            try {
                synchronous_cleanup(*data);
            } catch (...) {
                std::cerr << "Cleanup also failed during error handling" << std::endl;
            }
        }
        
        // 🔥 在任务真正完成后递减计数器
#ifdef TASK_COUNT
        int cur_task = --pending_task_count;
        cout << "now -1 has  " << cur_task << " tasks (memory copy completed)\n";
#endif
    });
#ifdef TASK_COUNT
        int cur_task = ++pending_task_count;
        cout << "now -1 has  " << cur_task << " tasks (memory copy completed)\n";
#endif
#ifdef DEBUG
    printf("[DEBUG] 📤 Memory copy task submitted to thread pool\n");
#endif
}

// 准备GPU数据阶段 (同步, 除开 内存复制)
void prepare_gpu_data_stage(AsyncGpuPipeline::AsyncTaskData& data) {
#ifdef DEBUG
    printf("[DEBUG] 📋 prepare_gpu_data_stage: Starting data preparation...\n");
#endif

    TaskManager& tm = data.task_manager;

    //1. 准备数据（类似原来的逻辑）
    Taskcontent h_tasks;

    data.result_len = 0;

    h_tasks.seg_types = tm.seg_types.data();
    h_tasks.seg_ids = tm.seg_ids.data();

    // ⚠️ 修复：保存字符串到data中
    data.all_prefixes = std::accumulate(tm.prefixs.begin(), tm.prefixs.end(), std::string(""));
    h_tasks.prefixs = data.all_prefixes.c_str();

#ifdef DEBUG
    printf("[DEBUG] 📋 Data preparation: taskcount=%d, guesscount=%d, prefixes_len=%zu\n",
           tm.taskcount, tm.guesscount, data.all_prefixes.length());
#endif

    data.h_prefix_offsets = new int[tm.prefixs.size() + 1];
    h_tasks.prefix_offsets = data.h_prefix_offsets;
    h_tasks.prefix_offsets[0] = 0;

    for (size_t i = 0; i < tm.prefixs.size(); ++i) {
        h_tasks.prefix_offsets[i + 1] = h_tasks.prefix_offsets[i] + tm.prefix_lens[i];
    }

    h_tasks.prefix_lens = tm.prefix_lens.data();
    h_tasks.taskcount = tm.taskcount;
    h_tasks.guesscount = tm.guesscount;
    h_tasks.seg_lens = tm.seg_lens.data();
    h_tasks.seg_value_counts = tm.seg_value_count.data();


    // ⚠️ 修复：保存偏移量到data中
    data.res_offset.clear();  // 清空而不是resize
    data.cumulative_offsets.resize(tm.taskcount + 1, 0);

    // 计算偏移量
    for(int i = 0; i < tm.taskcount; i++){
        data.res_offset.push_back(data.result_len);  // 这样就正确了
        data.result_len += tm.seg_value_count[i] * (tm.seg_lens[i] + tm.prefix_lens[i]);
        data.cumulative_offsets[i + 1] = data.cumulative_offsets[i] + tm.seg_value_count[i];
    }
    h_tasks.output_offsets = data.res_offset.data();
    h_tasks.cumulative_guess_offsets = data.cumulative_offsets.data();

    // 分配host buffer
    data.gpu_buffer = new char[data.result_len];

    //2. 分配GPU内存
    CUDA_CHECK(hipMalloc(&data.temp_prefixs, h_tasks.prefix_offsets[tm.prefixs.size()] * sizeof(char)));
    CUDA_CHECK(hipMalloc(&data.d_seg_types, tm.taskcount * sizeof(int)));
    CUDA_CHECK(hipMalloc(&data.d_seg_ids, tm.taskcount * sizeof(int)));
    CUDA_CHECK(hipMalloc(&data.d_seg_lens, tm.seg_lens.size() * sizeof(int)));
    CUDA_CHECK(hipMalloc(&data.d_prefix_offsets, (tm.prefixs.size() + 1) * sizeof(int)));
    CUDA_CHECK(hipMalloc(&data.d_prefix_lens, tm.prefix_lens.size() * sizeof(int)));
    CUDA_CHECK(hipMalloc(&data.d_seg_value_counts, tm.seg_value_count.size() * sizeof(int)));
    CUDA_CHECK(hipMalloc(&data.d_cumulative_guess_offsets, (tm.taskcount + 1) * sizeof(int)));
    CUDA_CHECK(hipMalloc(&data.d_output_offsets, (tm.taskcount + 1) * sizeof(int)));
    CUDA_CHECK(hipMalloc(&data.d_tasks, sizeof(Taskcontent)));
    CUDA_CHECK(hipMalloc(&data.d_guess_buffer, data.result_len * sizeof(char)));


    
    //3. 🔥 同步拷贝数据到GPU
    CUDA_CHECK(hipMemcpy(data.d_seg_types, h_tasks.seg_types, tm.taskcount * sizeof(int),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(data.d_seg_ids, h_tasks.seg_ids, tm.taskcount * sizeof(int),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(data.d_seg_lens, h_tasks.seg_lens, tm.seg_lens.size() * sizeof(int),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(data.temp_prefixs, h_tasks.prefixs, h_tasks.prefix_offsets[tm.prefixs.size()] * sizeof(char),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(data.d_prefix_offsets, h_tasks.prefix_offsets, (tm.prefixs.size() + 1) * sizeof(int),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(data.d_prefix_lens, h_tasks.prefix_lens, tm.prefixs.size() * sizeof(int),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(data.d_seg_value_counts, h_tasks.seg_value_counts, tm.seg_value_count.size() * sizeof(int),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(data.d_cumulative_guess_offsets, h_tasks.cumulative_guess_offsets, (tm.taskcount + 1) * sizeof(int),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(data.d_output_offsets, h_tasks.output_offsets, (tm.taskcount + 1) * sizeof(int),
                          hipMemcpyHostToDevice));



    // 准备Taskcontent结构
    data.task_content.seg_types = data.d_seg_types;
    data.task_content.seg_ids = data.d_seg_ids;
    data.task_content.seg_lens = data.d_seg_lens;
    data.task_content.prefixs = data.temp_prefixs;
    data.task_content.prefix_offsets = data.d_prefix_offsets;
    data.task_content.prefix_lens = data.d_prefix_lens;
    data.task_content.seg_value_counts = data.d_seg_value_counts;
    data.task_content.cumulative_guess_offsets = data.d_cumulative_guess_offsets;
    data.task_content.output_offsets = data.d_output_offsets;
    data.task_content.taskcount = tm.taskcount;
    data.task_content.guesscount = tm.guesscount;

    CUDA_CHECK(hipMemcpy(data.d_tasks, &data.task_content, sizeof(Taskcontent),
                          hipMemcpyHostToDevice));

#ifdef DEBUG
    printf("[DEBUG] 📋 prepare_gpu_data_stage: Data preparation completed, result_len=%zu\n", data.result_len);
#endif
}

// 启动kernel阶段（纯异步）
void launch_kernel_stage(AsyncGpuPipeline::AsyncTaskData& data) {
#ifdef DEBUG
    printf("[DEBUG] 🔥 launch_kernel_stage: Starting kernel launch...\n");
#endif

    // 🔥 第一步：检查所有关键变量是否存在
    printf("[DEBUG] 🔍 Checking all variables before kernel launch:\n");

    // 检查 gpu_data（全局变量）
    if (gpu_data == nullptr) {
        printf("[ERROR] ❌ gpu_data is NULL! This is likely the cause of 'invalid resource handle'\n");
        throw std::runtime_error("gpu_data is not initialized");
    } else {
        printf("[DEBUG] yes gpu_data is valid: %p\n", gpu_data);
    }

    // 检查 data 的关键GPU指针
    printf("[DEBUG] 🔍 Checking AsyncTaskData GPU pointers:\n");
    
    if (data.d_tasks == nullptr) {
        printf("[ERROR] ❌ data.d_tasks is NULL!\n");
        throw std::runtime_error("d_tasks is not allocated");
    } else {
        printf("[DEBUG] yes data.d_tasks is valid: %p\n", data.d_tasks);
    }

    if (data.d_guess_buffer == nullptr) {
        printf("[ERROR] ❌ data.d_guess_buffer is NULL!\n");
        throw std::runtime_error("d_guess_buffer is not allocated");
    } else {
        printf("[DEBUG] yes data.d_guess_buffer is valid: %p\n", data.d_guess_buffer);
    }

    if (data.compute_stream == nullptr) {
        printf("[ERROR] ❌ data.compute_stream is NULL!\n");
        throw std::runtime_error("compute_stream is not created");
    } else {
        printf("[DEBUG] yes data.compute_stream is valid: %p\n", data.compute_stream);
    }

    // 检查其他关键GPU指针
    printf("[DEBUG] 🔍 Checking other GPU memory allocations:\n");
    printf("[DEBUG] - temp_prefixs: %p %s\n", data.temp_prefixs, data.temp_prefixs ? "yes" : "no");
    printf("[DEBUG] - d_seg_types: %p %s\n", data.d_seg_types, data.d_seg_types ? "yes" : "no");
    printf("[DEBUG] - d_seg_ids: %p %s\n", data.d_seg_ids, data.d_seg_ids ? "yes" : "no");
    printf("[DEBUG] - d_seg_lens: %p %s\n", data.d_seg_lens, data.d_seg_lens ? "yes" : "no");
    printf("[DEBUG] - d_prefix_offsets: %p %s\n", data.d_prefix_offsets, data.d_prefix_offsets ? "yes" : "no");
    printf("[DEBUG] - d_prefix_lens: %p %s\n", data.d_prefix_lens, data.d_prefix_lens ? "yes" : "no");
    printf("[DEBUG] - d_seg_value_counts: %p %s\n", data.d_seg_value_counts, data.d_seg_value_counts ? "yes" : "no");
    printf("[DEBUG] - d_cumulative_guess_offsets: %p %s\n", data.d_cumulative_guess_offsets, data.d_cumulative_guess_offsets ? "yes" : "no");
    printf("[DEBUG] - d_output_offsets: %p %s\n", data.d_output_offsets, data.d_output_offsets ? "yes" : "no");

    // 检查TaskManager数据的有效性
    TaskManager& tm = data.task_manager;
    printf("[DEBUG] 🔍 Checking TaskManager data:\n");
    printf("[DEBUG] - taskcount: %d\n", tm.taskcount);
    printf("[DEBUG] - guesscount: %d\n", tm.guesscount);
    printf("[DEBUG] - seg_ids.size(): %zu\n", tm.seg_ids.size());
    printf("[DEBUG] - seg_lens.size(): %zu\n", tm.seg_lens.size());
    printf("[DEBUG] - prefixs.size(): %zu\n", tm.prefixs.size());
    printf("[DEBUG] - result_len: %zu\n", data.result_len);

    // 检查内核启动参数的合理性
    int total_threads_needed = (tm.guesscount + GUESS_PER_THREAD - 1) / GUESS_PER_THREAD;
    int threads_per_block = 1024;
    int blocks = (total_threads_needed + threads_per_block - 1) / threads_per_block;

    printf("[DEBUG] 🔍 Checking kernel launch parameters:\n");
    printf("[DEBUG] - GUESS_PER_THREAD: %d\n", GUESS_PER_THREAD);
    printf("[DEBUG] - total_threads_needed: %d\n", total_threads_needed);
    printf("[DEBUG] - threads_per_block: %d\n", threads_per_block);
    printf("[DEBUG] - blocks: %d\n", blocks);



    // 异步启动kernel
    generate_guesses_kernel<<<blocks, threads_per_block, 0, data.compute_stream>>>
        (gpu_data, data.d_tasks, data.d_guess_buffer);



    // 检查启动错误
    CUDA_CHECK(hipGetLastError());

#ifdef DEBUG
    printf("[DEBUG] 🔥 launch_kernel_stage: Kernel launched successfully\n");
#endif
}




// 处理字符串阶段
void process_strings_stage(AsyncGpuPipeline::AsyncTaskData& data) {

#ifdef DEBUG
    printf("[DEBUG] 🔍 process_strings_stage: Entry - thread_id=%zu\n", std::this_thread::get_id());
    fflush(stdout);
#endif
    TaskManager& tm = data.task_manager;

    // ⚠️ 检查错误状态
    if (data.has_error) {
#ifdef DEBUG
        printf("[DEBUG] ⚠️ process_strings_stage: Skipping due to error state\n");
        fflush(stdout);
#endif
        return;
    }


#ifdef DEBUG
printf("[DEBUG] 🔍 process_strings_stage: Processing %d segments, %d total guesses\n",
        (int)tm.seg_ids.size(), tm.guesscount);
printf("[DEBUG] 🔍 Buffer info: gpu_buffer=%p, result_len=%zu\n",
        data.gpu_buffer, data.result_len);
fflush(stdout);
#endif

    data.local_guesses.reserve(tm.guesscount);

#ifdef DEBUG
        printf("[DEBUG] 🔍 Reserved space for %d guesses\n", tm.guesscount);
        fflush(stdout);
#endif

    try {
        data.local_guesses.reserve(tm.guesscount);

#ifdef DEBUG
        printf("[DEBUG] 🔍 Reserved space for %d guesses\n", tm.guesscount);
        fflush(stdout);
#endif

        for (int i = 0; i < tm.seg_ids.size(); i++) {
#ifdef DEBUG
            if (i < 3) {  // 只打印前3个，避免输出过多
                printf("[DEBUG] 🔍 Processing segment %d: seg_value_count=%d, seg_len=%d, prefix_len=%d\n",
                       i, tm.seg_value_count[i], tm.seg_lens[i], tm.prefix_lens[i]);
                fflush(stdout);
            }
#endif

            for (int j = 0; j < tm.seg_value_count[i]; j++) {
                int start_offset = data.res_offset[i] + j * (tm.seg_lens[i] + tm.prefix_lens[i]);

#ifdef DEBUG
                if (i < 2 && j < 2) {  // 只打印前几个，避免输出过多
                    printf("[DEBUG] 🔍 Creating guess[%d][%d]: offset=%d, length=%d\n",
                           i, j, start_offset, tm.seg_lens[i] + tm.prefix_lens[i]);
                    fflush(stdout);
                }
#endif

                // 检查边界
                if (start_offset + tm.seg_lens[i] + tm.prefix_lens[i] > data.result_len) {
#ifdef DEBUG
                    printf("[DEBUG] ❌ Buffer overflow! offset=%d, length=%d, result_len=%zu\n",
                           start_offset, tm.seg_lens[i] + tm.prefix_lens[i], data.result_len);
                    fflush(stdout);
#endif
                    data.has_error = true;
                    return;
                }

                data.local_guesses.emplace_back(
                    data.gpu_buffer + start_offset,
                    tm.seg_lens[i] + tm.prefix_lens[i]
                );
            }
        }

#ifdef DEBUG
        printf("[DEBUG] ✅ process_strings_stage: Created %zu guesses successfully\n",
               data.local_guesses.size());
        fflush(stdout);
#endif

    } catch (const std::exception& e) {
#ifdef DEBUG
        printf("[DEBUG] ❌ process_strings_stage exception: %s\n", e.what());
        fflush(stdout);
#endif
        data.has_error = true;
    }

}

// 合并结果阶段
void merge_results_stage(AsyncGpuPipeline::AsyncTaskData& data) {
#ifdef DEBUG
    printf("[DEBUG] 🔗 merge_results_stage: Entry - thread_id=%zu\n", std::this_thread::get_id());
    fflush(stdout);
#endif
    // ⚠️ 检查错误状态
    if (data.has_error) {
#ifdef DEBUG
        printf("[DEBUG] ⚠️ merge_results_stage: Skipping due to error state\n");
        fflush(stdout);
#endif
        return;
    }


#ifdef DEBUG
    printf("[DEBUG] 🔗 merge_results_stage: Merging %zu guesses\n", data.local_guesses.size());
    fflush(stdout);
#endif


    try {
#ifdef DEBUG
        printf("[DEBUG] 🔗 Attempting to acquire locks...\n");
        fflush(stdout);
#endif

        {
            // std::lock_guard<std::mutex> lock1(main_data_mutex);

#ifdef DEBUG
            printf("[DEBUG] 🔗 main_data_mutex acquired\n");
            fflush(stdout);
#endif

            // std::lock_guard<std::mutex> lock2(gpu_buffer_mutex);
            std::scoped_lock lock(main_data_mutex, gpu_buffer_mutex);

#ifdef DEBUG
            printf("[DEBUG] 🔗 gpu_buffer_mutex acquired\n");
            printf("[DEBUG] 🔗 Current queue size: %zu\n", q.guesses.size());
            fflush(stdout);
#endif

            // 插入猜测结果到主队列
            q.guesses.insert(q.guesses.end(),
                             data.local_guesses.begin(),
                             data.local_guesses.end());

#ifdef DEBUG
            printf("[DEBUG] 🔗 Guesses inserted, new queue size: %zu\n", q.guesses.size());
            fflush(stdout);
#endif

            // 将GPU缓冲区指针加入管理列表
            if (data.gpu_buffer != nullptr) {
                pending_gpu_buffers.push_back(data.gpu_buffer);
                data.gpu_buffer = nullptr;
#ifdef DEBUG
                printf("[DEBUG] 🔗 GPU buffer added to pending list, total pending: %zu\n",
                       pending_gpu_buffers.size());
                fflush(stdout);
#endif
            } else {
                cout << " gpu_buffer ptr WRONG in merge_stage" << endl;
            }

#ifdef DEBUG
            printf("[DEBUG] 🔗 Releasing locks...\n");
            fflush(stdout);
#endif
        }

#ifdef DEBUG
        printf("[DEBUG] ✅ merge_results_stage: Completed successfully\n");
        fflush(stdout);
#endif

    } catch (const std::exception& e) {
#ifdef DEBUG
        printf("[DEBUG] ❌ merge_results_stage exception: %s\n", e.what());
        fflush(stdout);
#endif
        data.has_error = true;
    }
}


// 智能清理阶段（正常时异步，异常时同步）
void cleanup_stage(AsyncGpuPipeline::AsyncTaskData& data) {
#ifdef DEBUG
    printf("[DEBUG] 🧹 cleanup_stage: Starting resource cleanup (error=%s)...\n",
           data.has_error ? "true" : "false");
#endif

    if (data.has_error) {
        // 🚨 异常情况：懒得清理了。
#ifdef DEBUG
        printf("[DEBUG] ⚠️ Error detected in cleanup stage\n");
#endif
    } else {
        // ✅ 正常情况：异步清理，提高性能
#ifdef DEBUG
        printf("[DEBUG] ✅ Normal completion, performing asynchronous cleanup\n");
#endif
        synchronous_cleanup(data);
#ifdef DEBUG
        printf("[DEBUG] ✅ Normal completion,  asynchronous cleanup submitted\n");
#endif
    }
}


// 同步步清理（用于正常情况）
void synchronous_cleanup(AsyncGpuPipeline::AsyncTaskData& data) {
#ifdef DEBUG
    printf("[DEBUG] ⚡ synchronous_cleanup: Scheduling async GPU memory release...\n");
#endif

    // 🔥 使用异步释放，不阻塞当前线程
    // if (data.temp_prefixs) hipFreeAsync(data.temp_prefixs, data.compute_stream);
    // if (data.d_seg_types) hipFreeAsync(data.d_seg_types, data.compute_stream);
    // if (data.d_seg_ids) hipFreeAsync(data.d_seg_ids, data.compute_stream);
    // if (data.d_seg_lens) hipFreeAsync(data.d_seg_lens, data.compute_stream);
    // if (data.d_prefix_offsets) hipFreeAsync(data.d_prefix_offsets, data.compute_stream);
    // if (data.d_prefix_lens) hipFreeAsync(data.d_prefix_lens, data.compute_stream);
    // if (data.d_seg_value_counts) hipFreeAsync(data.d_seg_value_counts, data.compute_stream);
    // if (data.d_cumulative_guess_offsets) hipFreeAsync(data.d_cumulative_guess_offsets, data.compute_stream);
    // if (data.d_output_offsets) hipFreeAsync(data.d_output_offsets, data.compute_stream);
    // if (data.d_tasks) hipFreeAsync(data.d_tasks, data.compute_stream);
    // if (data.d_guess_buffer) hipFreeAsync(data.d_guess_buffer, data.compute_stream);
    // 🔥 同步释放GPU内存
    hipError_t sync_err = hipStreamSynchronize(data.compute_stream);
    if (sync_err != hipSuccess) {
        std::cerr << "Stream synchronization failed: " << hipGetErrorString(sync_err) << std::endl;
    }
#ifdef DEBUG
    printf("[DEBUG] ⚡ Stream synchronized, GPU operations completed\n");
#endif
    if (data.temp_prefixs)  hipFree(data.temp_prefixs);
    if (data.d_seg_types) hipFree(data.d_seg_types);
    if (data.d_seg_ids) hipFree(data.d_seg_ids);
    if (data.d_seg_lens) hipFree(data.d_seg_lens);
    if (data.d_prefix_offsets) hipFree(data.d_prefix_offsets);
    if (data.d_prefix_lens) hipFree(data.d_prefix_lens);
    if (data.d_seg_value_counts) hipFree(data.d_seg_value_counts);
    if (data.d_cumulative_guess_offsets) hipFree(data.d_cumulative_guess_offsets);
    if (data.d_output_offsets) hipFree(data.d_output_offsets);
    if (data.d_tasks) hipFree(data.d_tasks);
    if (data.d_guess_buffer) hipFree(data.d_guess_buffer);
    data.temp_prefixs = nullptr;
    data.d_seg_types = nullptr;
    data.d_seg_ids = nullptr;
    data.d_seg_lens = nullptr;
    data.d_prefix_offsets = nullptr;
    data.d_prefix_lens = nullptr;
    data.d_seg_value_counts = nullptr;
    data.d_cumulative_guess_offsets = nullptr;
    data.d_output_offsets = nullptr;
    data.d_tasks = nullptr;
    data.d_guess_buffer = nullptr;

    if (data.h_prefix_offsets) {
        delete[] data.h_prefix_offsets;
        data.h_prefix_offsets = nullptr;
    }
#ifdef DEBUG
    printf("[DEBUG] ✅ synchronous_cleanup: All resources cleaned\n");
#endif

#ifdef DEBUG
    cout << "begin delete asynctaskdata" << endl;
#endif
    //改成同步
    delete &data;


#ifdef DEBUG
    cout << "end delete asynctaskdata " << endl;

    if (data.gpu_buffer != nullptr) {
        cout << "Error, gpu_buffer ptr should be nullptr in async_cleanup" << endl;
    }
#endif


}



